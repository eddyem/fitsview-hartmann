#include "hip/hip_runtime.h"
/*
 * 		CUDA.cu - subroutines for GPU
 *
 *      Copyright 2011 Edward V. Emelianoff <eddy@sao.ru>
 *
 *      This program is free software; you can redistribute it and/or modify
 *      it under the terms of the GNU General Public License as published by
 *      the Free Software Foundation; either version 2 of the License, or
 *      (at your option) any later version.
 *
 *      This program is distributed in the hope that it will be useful,
 *      but WITHOUT ANY WARRANTY; without even the implied warranty of
 *      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *      GNU General Public License for more details.
 *
 *      You should have received a copy of the GNU General Public License
 *      along with this program; if not, write to the Free Software
 *      Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 *      MA 02110-1301, USA.
 */
#define _CUDA_CU_
#include "include/CUtools.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
//#include <hip/hip_runtime_api.h>
//#include <crt/device_runtime.h>
//#include <hip/device_functions.h>

const int SHMEMSZ = 16383; // default constants, changed runtime
const int QBLKSZ  = 16;		// QBLKSZ = sqrt(LBLKSZ)
const int LBLKSZ  = 512;

// static arrays for sines & cosines values
static float *Sin_d = NULL, *Cos_d = NULL;
// array size
static int sincosize = 0;

hipError_t CUerr;
inline int CUERROR(char *str){
	if(CUerr != hipSuccess){
		fprintf(stderr, "%s, %s\n", str, hipGetErrorString(CUerr));
		return 1;
	}else return 0;
}
// error macro (by default - nothing)
#define RETMACRO return
// memory macros
#define CUALLOC(var, size)		do{				\
	CUerr = hipMalloc((void**)&var, size);		\
	if(CUERROR("CUDA: can't allocate memory")){	\
		RETMACRO;								\
}}while(0)
#define CUMOV2DEV(dest, src, size) do{			\
	CUerr = hipMemcpy(dest, src, size,			\
				hipMemcpyHostToDevice);		\
	if(CUERROR("CUDA: can't copy data to device")){\
		RETMACRO;}								\
}while(0)
#define CUMOV2HOST(dest, src, size) do{			\
	CUerr = hipMemcpy(dest, src, size,			\
				hipMemcpyDeviceToHost);		\
	if(CUERROR("CUDA: can't copy data to host")){\
		RETMACRO;}								\
}while(0)
#define CUFREE(var) do{hipFree(var); var = NULL; }while(0)
#define  CUFFTCALL(fn)		do{					\
	hipfftResult fres = fn;						\
	if(HIPFFT_SUCCESS != fres){					\
		fprintf(stderr, "CUDA fft error %d\n", fres);\
		RETMACRO;}								\
}while(0)

#ifdef EBUG
	#define FNAME() fprintf(stderr, "\n%s (%s, line %d)\n", __func__, __FILE__, __LINE__)
	#define DBG(...) do{fprintf(stderr, "%s (%s, line %d): ", __func__, __FILE__, __LINE__); \
					fprintf(stderr, __VA_ARGS__);			\
					fprintf(stderr, "\n");} while(0)
#else
	#define FNAME()	 do{}while(0)
	#define DBG(...) do{}while(0)
#endif //EBUG

// getting the videocard parameters
extern "C" void getprops(){
	hipDeviceProp_t dP;
	hipDevice_t dev; hipCtx_t ctx;
	hipGetDeviceProperties(&dP, 0);
	hipDeviceGet(&dev,0);
	hipCtxCreate(&ctx, 0, dev);
	printf("\nDevice: %s, totalMem=%zd, memPerBlk=%zd,\n", dP.name, dP.totalGlobalMem, dP.sharedMemPerBlock);
	printf("warpSZ=%d, TPB=%d, TBDim=%dx%dx%d\n", dP.warpSize, dP.maxThreadsPerBlock,
			dP.maxThreadsDim[0],dP.maxThreadsDim[1],dP.maxThreadsDim[2]);
	printf("GridSz=%dx%dx%d, MemovrLap=%d, GPUs=%d\n", dP.maxGridSize[0],
			dP.maxGridSize[1],dP.maxGridSize[2],
			dP.deviceOverlap, dP.multiProcessorCount);
	printf("canMAPhostMEM=%d\n", dP.canMapHostMemory);
	printf("compute capability %d.%d.\n\n", dP.major, dP.minor);
	if(dP.major > 1){
	//	SHMEMSZ = 49151; QBLKSZ = 32; LBLKSZ = 1024;
	}
	size_t theFree, theTotal;
	hipError_t aaa = hipMemGetInfo( &theFree, &theTotal );
	printf("CARD returns(err=%d):  free mem:%zd,  total mem:%zd\n", aaa, theFree, theTotal);
	cuCtxDetach(ctx);
}

// normalisation of array arr with size arrsize
__global__ void normalize_vec(float *arr, int arrsize){
	__shared__ float max[LBLKSZ];
	int idx = threadIdx.x;
	int blksize = (arrsize + blockDim.x - 1) / blockDim.x;
	int b_beg = idx * blksize;
	if(b_beg >= arrsize) return;
	int b_end = b_beg + blksize;
	if(b_end > arrsize) b_end = arrsize;
	int i; float *ptr = &arr[b_beg];
	float mm = *ptr++;
	for(i = b_beg +1 ; i < b_end; i++, ptr++)
		if(mm < *ptr) mm = *ptr;
	max[idx] = mm;
	__syncthreads();
	if(idx == 0){
		mm = max[0];
		for(i = 1; i < LBLKSZ; i++)
			if(mm < max[i]) mm = max[i];
		max[0] = mm;
	}
	__syncthreads();
	ptr = &arr[b_beg];
	mm = max[0];
	if(mm != 0.f)
		for(i = b_beg ; i < b_end; i++, ptr++) *ptr /= mm;
}
/*
__global__ void fill_zeros(float *arr, int arrsize, int W, int H){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x > W || y > H) return;
	arr[x + y*W] = 0.f;
}*/

// kernel of function for sin/cos array initialisation
__global__ void fill_sincos(int angles,
							float *Sin_d, float *Cos_d,
							float anglestep, float conv){
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if(k >= angles) return;
	float theta = ((float)k / anglestep - 90.f) * conv;
	sincosf(theta, &Sin_d[k], &Cos_d[k]);
}

// initialisation of sin/cos arrays
extern "C" int init_sincos(int angles){
	#undef RETMACRO
	#define RETMACRO return 0
	// the value reciprocal for angle step
	float anglestep = (float)angles / 270.f;
	float conv = M_PI / 180.f;
	int blks = (angles + QBLKSZ - 1) / QBLKSZ;
	int threads = LBLKSZ;
	// first time we initialize arrays
	if(!Sin_d || !Cos_d || angles != sincosize){
		CUFREE(Cos_d);
		CUFREE(Sin_d);
		CUALLOC(Sin_d, angles*sizeof(float));
		CUALLOC(Cos_d, angles*sizeof(float));
		fill_sincos<<<blks, threads>>>(angles, Sin_d, Cos_d, anglestep, conv);
		sincosize = angles;
	}
	hipDeviceSynchronize();
	return 1;
	#undef RETMACRO
	#define RETMACRO return
}

/*
 * Lines Hough transform kernel
 * ima_d - device array with the image
 * imW, imH, min, max - width, height of the image and extreme values of its histogram
 * Sin_d, Cos_d - device array with sines and cosines of angles (-90..180degr increments 270/angles)
 * Rmax - the maximum range for the desired lines
 * angles - the number of angles in the range -90 .. 180
 * treshold - lower threshold of intensity (in relative units: I=tres*(max-min)+min) for inclusion of point into array
 * hough_d - output array with Hough transform
 */
__global__ void fill_lin_hough_array(float *ima_d,
						int imW, int imH,
						float min, float max,
						float *Sin_d, float *Cos_d,
						int Rmax, int angles, float treshold,
						float *hough_d){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int i = xi + imW * yi;
	float x = (float)xi;
	float y = (float)yi;
	int k, R;
	if(xi >= imW || yi >= imH) return;
	float wd = max-min; if(wd == 0.f) wd = 1.f;
	float ima = (ima_d[i]-min)/wd;
	if(ima > treshold){
		for(k = 0; k < angles; k++){
			// R = x*cos(theta) + y*sin(theta)
			R = (int)(0.5f + x * Cos_d[k] + y * Sin_d[k]);
			// THIS IS VERY BAD, BUT atomicAdd doesn't work in old devices
			if(R > 0 && R < Rmax) hough_d[R + Rmax*k] += ima;
			//if(R > 0 && R < Rmax) atomicAdd(&hough_d[R + Rmax*k], ima);
		}
	}
}

/*
 * Build Hough transform to find lines
 * Input:
 *		ima - the image data
 *		min, max - range of the data in it
 *		imW, imH - image width and height
 *		Rmax - the maximum value of R
 *		Angles - the array size of angles (the angle of pitch is 180/angles degrees)
 * Output:
 *		hough - array initialized by an external function,
 *		in which the Hough transform will be
 *		!!! array must be initialized with zeros before calling this function
 * Output array is normalized to unity
 */
extern "C" int fill_hough_lines(float *ima, float min, float max, int imW, int imH,
								int Rmax, int angles, float *hough){
	#undef RETMACRO
	#define RETMACRO do{ ret = 0; goto free_all; }while(0)
	int sz, ret = 1;
	int lblksz = LBLKSZ;
	float *ima_d = NULL, *hough_d = NULL;
	float treshold = 0.1f;
	sz = imW * imH;
	getprops();
	dim3 blkdim(QBLKSZ, QBLKSZ);
	dim3 griddim((imW+QBLKSZ-1)/QBLKSZ, (imH+QBLKSZ-1)/QBLKSZ);
//	dim3 hgriddim((Rmax+QBLKSZ-1)/QBLKSZ, (angles+QBLKSZ-1)/QBLKSZ);
	if(!init_sincos(angles)) RETMACRO;
	CUALLOC(ima_d, sz*sizeof(float));
	CUMOV2DEV(ima_d, ima, sz*sizeof(float));
	sz = Rmax * angles;
	CUALLOC(hough_d, sz*sizeof(float));
	hipMemset(hough_d, 0, sz*sizeof(float));
//	fill_zeros<<<hgriddim, blkdim>>>(hough_d, sz, Rmax, angles);
	hipDeviceSynchronize();
	CUMOV2DEV(hough_d, hough, sz*sizeof(float));
	fill_lin_hough_array<<<griddim, blkdim>>>(ima_d, imW,imH, min,max, Sin_d,Cos_d,
									Rmax, angles, treshold, hough_d);
	hipDeviceSynchronize();
	normalize_vec<<<1, lblksz>>>(hough_d, sz);
	hipDeviceSynchronize();
	CUMOV2HOST(hough, hough_d, sz*sizeof(float));
free_all:
	CUFREE(hough_d);
	CUFREE(ima_d);
	return ret;
	#undef RETMACRO
	#define RETMACRO return
}

/*
 * Kernels of the threshold filtering
 * in, out - in and out
 * stepfn - a pointer to a function of conversion
 * sizex, sizey - image size
 * min - the minimum intensity
 * wd - range of the data
 * step - a step for stepfn
 */
// uniform intensity distribution
__global__ void Funiform(float *in, int sizex, int sizey, float min, float step){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int i = xi + yi*sizex;
	if(xi >= sizex || yi >= sizey) return;
	in[i] = floor((in[i]-min)/step);
}
// logarithm distribution
__global__ void Flog(float *in, int sizex, int sizey, float min, float step){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int i = xi + yi*sizex;
	if(xi >= sizex || yi >= sizey) return;
	in[i] = floor(logf(in[i]-min+1.f)/step);
}
// exponential distribution
__global__ void Fexp(float *in, int sizex, int sizey, float min, float wd, float step){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int i = xi + yi*sizex;
	if(xi >= sizex || yi >= sizey) return;
	in[i] = floor(expf((in[i]-min)/wd)/step);
}
// distribution of a square root
__global__ void Fsqrt(float *in, int sizex, int sizey, float min, float step){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int i = xi + yi*sizex;
	if(xi >= sizex || yi >= sizey) return;
	in[i] = floor(sqrtf(in[i]-min)/step);
}
// distribution of a x^2
__global__ void Fpow(float *in, int sizex, int sizey, float min, float step){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int i = xi + yi*sizex;
	if(xi >= sizex || yi >= sizey) return;
	in[i] = floor((in[i]-min)*(in[i]-min)/step);
}

// functions for calculation of output scale
float Suniform(float in, float min, float wd, float step){
	return step*in + min;
}
float Slog(float in, float min, float wd, float step){
	return expf(in*step) + min - 1.f;
}
float Sexp(float in, float min, float wd, float step){
	return wd*logf(in*step) + min;
}
float Ssqrt(float in, float min, float wd, float step){
	return in*in*step*step + min;
}
float Spow(float in, float min, float wd, float step){
	return sqrtf(in*step) + min;
}

/*
 * Threshold filtering ("posterization")
 * Input:
 *		ima - picture (free() must be executed in the caller)
 *		f - filter:
 *			f-> w - number of levels of posterization, [2.255]
 *			f-> h - type of posterization (0 - uniform)
 *		sizex, sizey - image size
 *		min, max - minimum and maximum intensity of the image
 * Output:
 *		result - filtered image, the memory is allocated in this procedure
 *		scale - the scale of intensities, the memory is allocated here (if the scale!=NULL)
 *
 * TODO: save the result in the char, not float; learn display function
 */
extern "C"  int StepFilter(float *ima, float **result,
							Filter *f, int sizex, int sizey,
							float min, float max, float **scale){
	#undef RETMACRO
	#define RETMACRO do{ ret = 0; goto free_all; }while(0)
	int ret = 1;
	float wd = max - min;
	int y;
	float Nsteps = (float)f->w; // number of intervals
	float step;
	float *in=NULL; // image and result array
	int sz = sizex*sizey*sizeof(float);
	dim3 blkdim(QBLKSZ, QBLKSZ);
	dim3 griddim((sizex+QBLKSZ-1)/QBLKSZ, (sizey+QBLKSZ-1)/QBLKSZ);
	*result = (float*)malloc(sz);
	if(!result) RETMACRO;
	float (*scalefn)(float,float,float,float);
	if(f->w < 2 || f->w > 255) return 0;
	if(wd == 0.f) return 0;
	CUALLOC(in, sz);
	CUMOV2DEV(in, ima, sz);
	switch(f->h){ // filter type
		case LOG: // logarithm
			scalefn = Slog;
			step = logf(max-min+1.f)/Nsteps;
			Flog<<<griddim, blkdim>>>(in, sizex, sizey, min, step);
		break;
		case EXP: // exponential
			scalefn = Sexp;
			step = expf(1.f)/Nsteps;
			Fexp<<<griddim, blkdim>>>(in, sizex, sizey, min, wd, step);
		break;
		case SQRT: // square root
			scalefn = Ssqrt;
			step = sqrtf(wd)/Nsteps;
			Fsqrt<<<griddim, blkdim>>>(in, sizex, sizey, min, step);
		break;
		case POW: // power of two
			scalefn = Spow;
			step = wd*wd/Nsteps;
			Fpow<<<griddim, blkdim>>>(in, sizex, sizey, min, step);
		break;
		default: // uniform
			scalefn = Suniform;
			step = wd/Nsteps;
			Funiform<<<griddim, blkdim>>>(in, sizex, sizey, min, step);
	}
	hipDeviceSynchronize();
	CUMOV2HOST(*result, in, sz);
	if(scale){
		int M = f->w;
		*scale = (float*)calloc(M, sizeof(float));
		if(*scale) for(y = 0; y < M; y++){
			(*scale)[y] = scalefn(y+1,min,wd,step);
		}
	}
free_all:
	CUFREE(in);
	return ret;
	#undef RETMACRO
	#define RETMACRO return
}


/*
 * A set of functions for constructing differential filters
 */
int p2oi(int i){
	unsigned int v = (unsigned int)i - 1;
	v |= v >> 1;
	v |= v >> 2;
	v |= v >> 4;
	v |= v >> 8;
	v |= v >> 16;
	v++;
	return (int) v;
}
int nextpow2(int i, int j){
	int p1 = p2oi(i), p2 = p2oi(j);
	return (p1 > p2)? p1 : p2;
}
// multiplication of two complex matrices with size x size
// result in the entry of the first matrix
__global__ void ComplexMul(hipfftComplex *inout, hipfftComplex *in, int size){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	if(xi >= size || yi >= size) return;
	int i = xi + yi*size;
	hipfftComplex a = inout[i], b = in[i];
	inout[i].x = a.x * b.x - a.y * b.y;
	inout[i].y = a.x * b.y + a.y * b.x;
}

// restore coordinates of the Fourier transform
__global__ void fftshift(int size, float *m){
	int h = size/2;
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	if(xi >= h || yi >= h) return;
	// k - point in left upper quadrant, k1 - in right upper
	int k = yi * size+xi, k1 = k + h;
	// p - point in right lower quadrant, p1 - in left lower
	int p = k + (size+1)*h, p1 = k1 + (size-1)*h;
	float tmp;
	tmp = m[k]; m[k] = m[p]; m[p] = tmp;
	tmp = m[k1]; m[k1] = m[p1]; m[p1] = tmp;
}
// data copying float->hipfftReal (need because different sizes of picture and Fourier image)
__global__ void f2r(hipfftReal *out, float *in, int sizex, int sizey, int size2){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	if(xi >= sizex || yi >= sizey) return;
	out[xi+yi*size2] = (hipfftReal) in[xi+yi*sizex];
}
// data copying hipfftReal->float
__global__ void r2f(float *out, hipfftReal *in, int sizex, int sizey, int size2){
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	if(xi >= sizex || yi >= sizey) return;
	out[xi+yi*sizex] = (float) in[xi+yi*size2];
}

/*
 * The kernel of the Laplacian of Gaussian
 * Output:
 *		mask - the filled array
 * Input:
 *		size - array size (size x size)
 *		x0, x1 - array bounds on x: [x0, x1)  (outside this array filled by zeros)
 *		y0, y1 - -//- on y
 *		half - half the size of the array
 *		ss - normalizing factor
 *		sx2, sy2 - the variance of the filter in x and y
 */
__global__ void LGf_kernel(hipfftReal *mask, int size, int x0, int x1,
							int y0, int y1, float half, float ss,
							float sx2, float sy2){
	int xi = blockIdx.x * blockDim.x + threadIdx.x + x0;
	int yi = blockIdx.y * blockDim.y + threadIdx.y + y0;
	if(xi >= x1 || yi >= y1) return;
	int i = xi + yi*size;
	float x2 = (float)xi + half;
	float y2 = (float)yi + half;
	x2 = x2*x2 / sx2; y2 = y2*y2 / sy2;
	mask[i] = (hipfftReal)(ss * ((x2-1.f)/sx2+(y2-1.f)/sy2)*expf(-(x2+y2)/2.f));
}
// The kernel of Gaussian filter
__global__ void Gf_kernel(hipfftReal *mask, int size, int x0, int x1,
							int y0, int y1, float half, float ss,
							float sx2, float sy2){
	int xi = blockIdx.x * blockDim.x + threadIdx.x + x0;
	int yi = blockIdx.y * blockDim.y + threadIdx.y + y0;
	if(xi >= x1 || yi >= y1) return;
	int i = xi + yi*size;
	float x2 = (float)xi + half;
	float y2 = (float)yi + half;
	x2 = x2*x2 / sx2; y2 = y2*y2 / sy2;
	mask[i] = (hipfftReal)(ss * expf(-(x2+y2)/2.f));
}
/*
 * Building mask of Gaussian or Laplasian of Gaussian
 * Output:
 *		mask - filter array
 * Input:
 * 		size - mask size (size x size)
 * 		f - filter parameters
 */
void build_GLG_filter(hipfftReal *mask, int size, Filter *f){
	int y0=0,y1=size, x0=0, x1=size;
	float sx2 = f->sx * f->sx, sy2 = f->sy * f->sy;
	float half;
	dim3 blkdim(QBLKSZ, QBLKSZ);
	dim3 griddim((size+QBLKSZ-1)/QBLKSZ, (size+QBLKSZ-1)/QBLKSZ);
	if(f->w < size && f->w > 0){
		x0 = (size - f->w + 1) / 2;
		x1 = x0 + f->w;
	}
	if(f->h < size && f->h > 0){
		y0 = (size - f->h + 1) / 2;
		y1 = y0 + f->h;
	}
	half = -(float)size / 2.f;
	float ss = 3.f / half / half / sqrt(-half);
	switch(f->FilterType){
		case LAPGAUSS:
			LGf_kernel<<<griddim, blkdim>>>(mask, size, x0,x1, y0,y1, half, ss, sx2, sy2);
		break;
		case GAUSS:
			Gf_kernel<<<griddim, blkdim>>>(mask, size, x0,x1, y0,y1, half, ss, sx2, sy2);
		break;
		default:
			fprintf(stderr, "Error: bad filter\n");
	}
	hipDeviceSynchronize();
	DBG("size=%d, x0=%d,x1=%d, y0=%d,y1=%d, half=%g, ss=%g, sx2=%g, sy2=%g",
		size, x0,x1, y0,y1, half, ss, sx2, sy2);
}

/*
 * Building of elementary filter mask
 * Output:
 *		mask - filter array
 * Input:
 * 		size - mask size (size x size)
 * 		f - filter parameters
 */
void build_S_filter(hipfftReal *mask, int size, Filter *f){
	int y, a0, a1;
	float hh, Y, pt = 0.f;
	a0 = (size - 2) / 2;
	a1 = a0 + 3;
	hh = -(float)(size / 2);
	float ss = 1.f / (M_PI*2.f) / hh / hh / sqrtf(-hh);
	Y = -1.f;
	for(y = a0; y < a1; y++, Y+=1.f){
		float X = -1.f;
		int str, x;
		str = y * size;
		for(x = a0; x < a1; x++, X+=1.f){
			switch(f->FilterType){
				case SOBELH:
					pt = -X*(2.f-fabs(Y));
				break;
				case SOBELV:
					pt = -Y*(2.f-fabs(X));
				break;
				case PREWITTH:
					pt = X;
				break;
				case PREWITTV:
					pt = Y;
				break;
			}
			hipfftReal tmppar = (hipfftReal)ss*pt;
			hipMemcpy(&mask[str + x], &tmppar, sizeof(hipfftReal), hipMemcpyHostToDevice);
		}
	}
}
/*
 * Convolution filtering (convolution by FFT)
 * Input:
 * 		ima - picture, that need to be filtering
 * 		f - filter parameters
 * Output:
 * 		result - memory area, allocated by this function,
 * 				where the filtered picture to be store
 * return: TRUE if the filtering succeed
 */
extern "C"  int DiffFilter(float *ima, float **result,
					Filter *f, int sizex, int sizey){
	#undef RETMACRO
	#define RETMACRO do{ ret = 0; goto free_all; }while(0)
	int ssize, ret = 0, size2;
	float *tmp;
	size2 = nextpow2(sizex, sizey);
	ssize = size2 * size2; // Fourier image size
	dim3 blkdim(QBLKSZ, QBLKSZ);
	dim3 griddim((size2+QBLKSZ-1)/QBLKSZ, (size2+QBLKSZ-1)/QBLKSZ);
	dim3 halfgriddim((size2/2+QBLKSZ-1)/QBLKSZ, (size2/2+QBLKSZ-1)/QBLKSZ);
	dim3 imgriddim((sizex+QBLKSZ-1)/QBLKSZ, (sizey+QBLKSZ-1)/QBLKSZ);
	if(!result || !*result || !ima || !f){
		fprintf(stderr, "DiffFilter: bad parameters\n");
		return 0;
	}
	hipfftHandle plan;
	hipfftComplex *Fmask=NULL, *Fimg=NULL;
	hipfftReal *mask=NULL, *img=NULL, *resm=NULL;
	#ifdef EBUG
	getprops();
	#endif
	// Allocate memory for new objects
	DBG("allocate");
	CUALLOC(img, ssize*sizeof(hipfftReal));
	// fill it zeros
	hipMemset(img, 0, ssize*sizeof(hipfftReal));
	// copy ima -> img
	DBG("copy image to dev");
	CUALLOC(tmp, sizex*sizey*sizeof(float));
	CUMOV2DEV(tmp, ima, sizex*sizey*sizeof(float));
	f2r<<<imgriddim, blkdim>>>(img, tmp, sizex, sizey, size2);
	hipDeviceSynchronize();
	CUFREE(tmp);
	CUALLOC(Fimg, ssize*sizeof(hipfftComplex));
	// make FFT
	DBG("doing image FFT");
	CUFFTCALL(hipfftPlan2d(&plan, size2, size2, HIPFFT_R2C));
	CUFFTCALL(hipfftExecR2C(plan, img, Fimg));
	CUFREE(img);
	DBG("allocate");
	CUALLOC(mask, ssize*sizeof(hipfftReal));
	hipMemset(mask, 0, ssize*sizeof(hipfftReal));
	CUALLOC(Fmask, ssize*sizeof(hipfftComplex));
	switch(f->FilterType){
		case LAPGAUSS:
		case GAUSS:
			build_GLG_filter(mask, size2, f);
			break;
		case SOBELH:
		case SOBELV:
		case PREWITTH:
		case PREWITTV:
			build_S_filter(mask, size2, f);
			break;
		default:
			fprintf(stderr, "Error: bad filter\n");
			RETMACRO;
	}
	// swap filter quadrants
	fftshift<<<halfgriddim, blkdim>>>(size2, mask);
	hipDeviceSynchronize();
	// make FFT
	DBG("doing filter FFT");
	CUFFTCALL(hipfftExecR2C(plan, mask, Fmask));
	CUFFTCALL(hipfftDestroy(plan));
	CUFREE(mask);
	// make convolution in Fourier space
	DBG("multiplication");
	ComplexMul<<<griddim, blkdim>>>(Fimg, Fmask, size2);
	hipDeviceSynchronize();
	CUFREE(Fmask);
	// Inverse FFT
	DBG("doing inverse FFT");
	CUALLOC(resm, ssize*sizeof(hipfftReal));
	CUFFTCALL(hipfftPlan2d(&plan, size2, size2, HIPFFT_C2R));
	CUFFTCALL(hipfftExecC2R(plan, Fimg, resm));
	CUFFTCALL(hipfftDestroy(plan));
	CUFREE(Fimg);
	DBG("allocate");
	CUALLOC(tmp, ssize*sizeof(float));
	*result = (float*)calloc(sizex*sizey, sizeof(float));
	if(!*result) RETMACRO;
	// copy iFFT -> res
	DBG("copy to host");
	r2f<<<imgriddim, blkdim>>>(tmp, resm, sizex, sizey, size2);
	hipDeviceSynchronize();
	CUMOV2HOST(*result, tmp, sizex*sizey*sizeof(float));
	ret = 1;
free_all:
	CUFREE(Fmask); CUFREE(Fimg); CUFREE(img);
	CUFREE(mask); CUFREE(resm); CUFREE(tmp);
	#ifdef EBUG
	getprops();
	#endif
	return ret;
	#undef RETMACRO
	#define RETMACRO return
}
extern "C"  int MedFilter(float *ima, float **result, Filter *f, int sizex, int sizey){return 0;}
extern "C"  int GradFilterSimple(float *ima, float **result, Filter *f, int sizex, int sizey){return 0;}
